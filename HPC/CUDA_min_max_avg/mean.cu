#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

// CUDA kernel. Each thread takes care of one element of c
__global__ void vecAdd(double *a, double *b, double *c, int n)
{
    // Get our global thread ID
    int id = blockIdx.x*blockDim.x+threadIdx.x;			// get global index

    // Make sure we do not go out of bounds
    if (id < n)
        c[id] = a[id] + b[id];
}

int main( )
{

    //int n = 100000;
	int n=5;			// Size of vectors


    double *h_a;		// Host input vector
    double *h_b;		// Host input vector


    double *h_c;		//Host output vector


    double *d_a;		// Device input vector
    double *d_b;		// Device input vector

    double *d_c;		 //Device output vector


    size_t bytes = n*sizeof(double);		 // Size, in bytes, of each vector

    // Allocate memory for each vector on host
    h_a = (double*)malloc(bytes);
    h_b = (double*)malloc(bytes);
    h_c = (double*)malloc(bytes);

    // Allocate memory for each vector on GPU
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    int i;

    // Initialize vectors on host
    for( i = 0; i < n; i++ ) {

	h_a[i]=i;
	h_b[i]=i;

    }

    // Copy host vectors to device
    hipMemcpy( d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy( d_b, h_b, bytes, hipMemcpyHostToDevice);

    int blockSize, gridSize;

    // Number of threads in each thread block
    blockSize = 1024;

    // Number of thread blocks in grid
    gridSize = (int)ceil((float)n/blockSize);

    // Execute the kernel
    vecAdd<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);

    // Copy array back to host
    hipMemcpy(h_c,d_c, bytes, hipMemcpyDeviceToHost );

    // Sum up vector c and print result divided by n, this should equal 1 within error
    double sum = 0;
//    for(i=0;i<n;i++)
//    	printf("%f ",h_c[i]);
    for(i=0; i<n; i++){
        sum += h_c[i];
    }
    printf("Average mean of 2 vectors: %f\n", sum/n);

    // Release device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Release host memory
    free(h_a);
    free(h_b);
    free(h_c);

    return 0;
}
